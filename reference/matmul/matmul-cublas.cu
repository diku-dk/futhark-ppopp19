// Based on https://stackoverflow.com/a/23743838/6131552

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <assert.h>
#include <sys/time.h>

#define cudaCheckErrors(msg)                                    \
  do {                                                          \
    hipError_t __err = hipGetLastError();                     \
    if (__err != hipSuccess) {                                 \
      fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n",        \
              msg, hipGetErrorString(__err),                   \
              __FILE__, __LINE__);                              \
      fprintf(stderr, "*** FAILED - ABORTING\n");               \
      exit(1);                                                  \
    }                                                           \
  } while (0)


int GPU_Single(float *h_M, float *h_N, float *h_P, size_t ROWM, size_t COLM, size_t COLN, float alpha, float beta)
{

  float *d_M;
  float *d_N;
  float *d_P;

  size_t N_size =sizeof(float) *ROWM*COLM;
  size_t M_size =sizeof(float) *COLM*COLN;
  size_t P_size =sizeof(float) *ROWM*COLN;

  hipblasHandle_t myhandle;
  hipblasStatus_t cublas_result;

  hipMalloc(&d_M, M_size);
  hipMalloc(&d_N, N_size);
  hipMalloc(&d_P, P_size);
  cudaCheckErrors("hipMalloc fail");

  hipMemcpy(d_M, h_M, M_size , hipMemcpyHostToDevice);
  hipMemcpy(d_N, h_N, N_size , hipMemcpyHostToDevice);
  hipMemcpy(d_P, h_P, P_size , hipMemcpyHostToDevice);
  cudaCheckErrors("hipMemcpy H2D fail");

  cublas_result = hipblasCreate(&myhandle);
  assert(cublas_result == HIPBLAS_STATUS_SUCCESS);

  struct timeval t_start, t_end;
  gettimeofday(&t_start, NULL);
  int runtime, runs = 10;
  for (int i = 0; i < runs; i++) {
    cublas_result = hipblasSgemm(myhandle, HIPBLAS_OP_N, HIPBLAS_OP_N, ROWM, COLN, COLM, &alpha, d_M, ROWM, d_N, COLM, &beta, d_P, ROWM);
    hipDeviceSynchronize();
  }
  gettimeofday(&t_end, NULL);
  assert(cublas_result == HIPBLAS_STATUS_SUCCESS);

  runtime = ((t_end.tv_sec*1000000+t_end.tv_usec) - (t_start.tv_sec*1000000+t_start.tv_usec))/runs;

  hipMemcpy(h_P, d_P, P_size, hipMemcpyDeviceToHost);
  hipFree(d_M);
  hipFree(d_N);
  hipFree(d_P);
  cudaCheckErrors("hipMemcpy D2H fail");

  return runtime;
}

int main(int argc, char** argv) {
  if (argc != 2 && ((argc-2)%3!=0)) {
    fprintf(stderr, "%s usage: <outfile> < <n> <m> <k> >...\n", argv[0]);
    exit(1);
  }

  const char *outfile = argv[1];
  FILE *f = fopen(outfile, "w");
  assert(f != NULL);

  // For simplicity of plotting, we create a JSON file similar to what
  // futhark-bench would produce.
  fprintf(f, "{\"benchmarks/matmul.fut\":{\"datasets\":{\n");

  for (int i = 2; i < argc; i += 3) {
    int n = atoi(argv[i]);
    int m = atoi(argv[i+1]);
    int k = atoi(argv[i+2]);

    for (int x = n; x <= m; x++) {
      int y = k - (x+x);
      int ROWM = 1 << x;
      int COLM = 1 << y;
      int COLN = 1 << x;

      float *h_M1 = (float*) malloc(ROWM*COLM*sizeof(float));
      float *h_N1 = (float*) malloc(COLM*COLN*sizeof(float));
      float *h_P1 = (float*) malloc(ROWM*COLN*sizeof(float));

      printf("Multiplying [2**%d][2**%d] and [2**%d][2**%d] matrices\n", x, y, y, x);
      int runtime = GPU_Single(h_M1, h_N1, h_P1, ROWM, COLM, COLN, 1.0f, 0.0f);
      printf("Runtime in microseconds based on %d runs:\n%d\n",
             10, runtime);

      if (x != n || i != 2) {
        fprintf(f, ", ");
      } else {
        fprintf(f, "  ");
      }
      fprintf(f, "\"matmul-data/2pow%d_work_2pow%d_outer\":{\"runtimes\": [%d]}\n", k, x, runtime);

      free(h_M1);
      free(h_N1);
      free(h_P1);
    }
  }

  fprintf(f, "}}}\n");
  fclose(f);

  return 0;
}
